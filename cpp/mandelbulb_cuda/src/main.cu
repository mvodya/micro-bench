#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <fstream>
#include <iostream>
#include <microbench.hpp>

#include "mandelbulb_config.h"

using namespace MicroBench;

// Image width ? height in pixels
const uint16_t WIDTH = MB_MANDELBULB_RESOLUTION_WIDTH, HEIGHT = MB_MANDELBULB_RESOLUTION_HEIGHT;

// Maximum number of steps for ray marching
__device__ const int MAX_MARCHING_STEPS = MB_MANDELBULB_MAX_MARCHING_STEPS;
// Maximum marching distance before stopping
__device__ const double MAX_DISTANCE = MB_MANDELBULB_MAX_DISTANCE;
// Minimum distance threshold to consider on surface
__device__ const double SURFACE_DISTANCE = MB_MANDELBULB_SURFACE_DISTANCE;

// Maximum iterations for distance estimator function
__device__ const int MAX_ITERATIONS = MB_MANDELBULB_MAX_ITERATIONS;
// Escape radius for distance estimator
__device__ const double BAILOUT = MB_MANDELBULB_BAILOUT;
// Power exponent for Mandelbulb fractal shape
__device__ const double POWER = MB_MANDELBULB_POWER;

__device__ const double EPS = 0.001;

// Custom CVec3 class for CUDA
struct CVec3 {
  // Vector values
  double x, y, z;

  __device__ CVec3(double x = 0, double y = 0, double z = 0)
      : x(x), y(y), z(z) {}

  // Summation
  __device__ CVec3 operator+(const CVec3& v) const {
    return CVec3(x + v.x, y + v.y, z + v.z);
  }
  // Subtraction
  __device__ CVec3 operator-(const CVec3& v) const {
    return CVec3(x - v.x, y - v.y, z - v.z);
  }
  // Multiplication by value
  __device__ CVec3 operator*(double s) const {
    return CVec3(x * s, y * s, z * s);
  }
  // Scalar multiplication
  __device__ double operator*(const CVec3& v) const {
    return x * v.x + y * v.y + z * v.z;
  }
  // Calculate vector length
  __device__ double length() const { return sqrt(x * x + y * y + z * z); }
  // Get vector normalize
  __device__ CVec3 normalize() const {
    double len = sqrt(x * x + y * y + z * z);
    return CVec3(x / len, y / len, z / len);
  }
};

// Distance estimator (DE) function for the Mandelbulb fractal
// Calculate minimum distance from point to fractal
__device__ double mandelbulbDE(const CVec3& pos) {
  // Position
  CVec3 z = pos;
  // Distance increment
  double dr = 1.0;
  // Distance
  double r = 0.0;

  for (int i = 0; i < MAX_ITERATIONS; ++i) {
    r = z.length();
    if (r > BAILOUT) break;

    // Convert to spherical coordinates
    double theta = acos(z.z / r);
    double phi = atan2(z.y, z.x);
    double zr = pow(r, POWER);

    // Scale the distance increment
    dr = pow(r, POWER - 1.0) * POWER * dr + 1.0;

    // Calculate new coordinates
    double sinTheta = sin(POWER * theta);
    double cosTheta = cos(POWER * theta);
    double sinPhi = sin(POWER * phi);
    double cosPhi = cos(POWER * phi);

    z = CVec3(zr * sinTheta * cosPhi, zr * sinTheta * sinPhi, zr * cosTheta);
    z = z + pos;
  }
  return 0.5 * log(r) * r / dr;
}

// Calculate the normal vector at a surface point
__device__ CVec3 getNormal(const CVec3& pos) {
  double dx = mandelbulbDE(pos + CVec3(EPS, 0, 0)) -
              mandelbulbDE(pos - CVec3(EPS, 0, 0));
  double dy = mandelbulbDE(pos + CVec3(0, EPS, 0)) -
              mandelbulbDE(pos - CVec3(0, EPS, 0));
  double dz = mandelbulbDE(pos + CVec3(0, 0, EPS)) -
              mandelbulbDE(pos - CVec3(0, 0, EPS));
  CVec3 normal = CVec3(dx, dy, dz);
  return normal.normalize();
}

// Ray marching function to find distance to the surface
__device__ double rayMarch(const CVec3& from, const CVec3& dir) {
  double distance = 0.0;
  for (int i = 0; i < MAX_MARCHING_STEPS; ++i) {
    CVec3 pos = from + dir * distance;
    double d = mandelbulbDE(pos);
    if (d < SURFACE_DISTANCE) {
      // Intersection
      return distance;
    }
    if (distance > MAX_DISTANCE) {
      // No intersection
      break;
    }
    distance += d;
  }
  // No intersection found
  return -1.0;
}

// CUDA kernel for Mandelbulb rendering
__global__ void mandelbulbKernel(uint8_t* buffer) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= WIDTH || y >= HEIGHT) return;

  int idx = (y * WIDTH + x) * 3;
  CVec3 cameraPos(0.0, 0.0, -1.6);
  CVec3 lightDir = CVec3(1.0, 1.0, -1.0).normalize();

  // Normalize pixel coordinates
  double px = (x / (double)WIDTH) * 2.0 - 1.0;
  double py = (y / (double)HEIGHT) * 2.0 - 1.0;
  // Adjust for aspect ratio
  px *= (double)WIDTH / (double)HEIGHT;

  // Calculate ray direction
  CVec3 dir = CVec3(px, py, 1.0).normalize();

  double dist = rayMarch(cameraPos, dir);
  if (dist > 0.0) {
    // Calculate intersection point and normal
    CVec3 pos = cameraPos + dir * dist;
    CVec3 normal = getNormal(pos);

    // Compute lighting (Lambertian shading)
    double diffuse = max(0.0, normal * lightDir);

    // Set pixel color based on lighting
    uint8_t color = (uint8_t)(diffuse * 255);
    buffer[idx] = color;
    buffer[idx + 1] = color;
    buffer[idx + 2] = color;
  } else {
    // Set background color
    buffer[idx] = buffer[idx + 1] = buffer[idx + 2] = 0;
  }
}

int main() {
  const size_t STEPS = MB_RENDER_NUM;
  std::cout << "[CPP CUDA] Mandelbulb GPU benchmark\n";
  std::cout << "Size: " << WIDTH << " x " << HEIGHT << "\nPOWER: " << POWER
            << " / MAX ITERATIONS: " << MAX_MARCHING_STEPS << "\nCUDA Block: 16 x 16 = 256\n";

  uint8_t* d_buffer;
  // Allocate buffer on GPU device
  hipMalloc(&d_buffer, WIDTH * HEIGHT * 3);

  // 16 per 16 threads = 256 threads in one block
  dim3 blockSize(16, 16);
  dim3 gridSize((WIDTH + blockSize.x - 1) / blockSize.x,
                (HEIGHT + blockSize.y - 1) / blockSize.y);

  // Steps
  for (size_t i = 0; i < STEPS; i++) {
    std::cout << "Start rendering " << i + 1 << "/" << STEPS << "\n";
    auto duration = measureExecutionTime([&]() {
      mandelbulbKernel<<<gridSize, blockSize>>>(d_buffer);
      hipDeviceSynchronize();
    });
    std::cout << "[" << i + 1 << "] Execution time: " << duration << " ns ("
              << (double)duration / 1000000000. << " s)\n";

    if (i == 0) {
      std::cout << "Save first run to file mandelbulb.ppm\n";
      Image* image = new Image(WIDTH, HEIGHT);
      hipMemcpy(image->getBuffer(), d_buffer, WIDTH * HEIGHT * 3,
                 hipMemcpyDeviceToHost);
      image->saveFile("mandelbulb.ppm");
      delete image;
    }
  }

  hipFree(d_buffer);
  std::cout << "All renderings are done\n";
  return 0;
}